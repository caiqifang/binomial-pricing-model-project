#include "hip/hip_runtime.h"
#include <algorithm>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>
#include <stdio.h>

#include "cudaModel.h"

/*
   This file is the implementation of parallel binomial model
 */

////////// KERNEL /////////////////////////
// In this project, we are calculating the call price
// This kernel function calculate the call option value based on strike price
__device__ __inline__ double getValue(double strike, double stock){
    double value = stock - strike;
    if(value > 0.0)
        return value;
    return 0.0;
}

// get the starting index in the array based on current level and define
__device__ __inline__ int getIndex(int length, int maxL, int level){
    return length*((maxL+1+level+2)*(maxL-level)/2);
}

// calculate the leaf level node value in the binomial tree
// finding intrinsic value of the financial product
__global__ void kernelFinalStage(int length, int maxL,  double strike,
        double* device_buf, double device_u, double device_d, double* device_s){
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    int size = maxL+1;
    if( index >= length * size)
        return; // output bound
    int element = index / size;
    double rank = __int2double_rn(index % size);
    double stock = device_s[element]; // s0
    //double up = device_u[element];
    //double down = device_d[element];
    double maxLevel = __int2double_rn(maxL);
    stock = stock * pow(device_u, maxLevel-rank) * pow(device_d, rank);
    device_buf[index] = getValue(strike, stock);
    return;
}

// parallel compute the value in all nodes in the same level,
// skiping the leaf level nodes
__global__ void kernelCalc(int level, int maxL, int length, double device_p,
        double device_q, double device_r, double* device_buf){
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    int size = level + 1;
    if( index >= length * size)
        return; // output bound
    // WATCH!!
    int curr_start = getIndex( length, maxL, level);
    int prev_start = getIndex( length, maxL, level+1);
    int element = index / size;
    int rank = index % size;
    //double p = device_p[element];
    //double q = device_q[element];
    int head = prev_start + element*(size+1) + rank;
    int tail = head + 1;
    device_buf[curr_start+index] = 1.0/(1+device_r)*(device_p*device_buf[head] +
            device_q*device_buf[tail]);
    return;
}

// parallel copy all outputs to the CPU output array
__global__ void kernelOutput(double* device_output, double* device_buf,
        int length, int maxL){
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    if (index >= length) return; /// out of bound
    int total_length = length*(maxL+2)*(maxL+1)/2;
    device_output[index] = device_buf[total_length - length + index];
    return;
}

//////////// HOST CODE ///////////////////////////
// initialize the buffer and related structure
CudaModel::CudaModel(double s){
    hipMalloc(&device_buf, sizeof(double)*MAXLENGTH*BUFSIZE);
    hipMalloc(&device_output, sizeof(double)*MAXLENGTH);
    hipMalloc(&device_s, sizeof(double)*MAXLENGTH);
    strike = s;
}

// class destructor
CudaModel::~CudaModel(){
    hipFree(device_buf);
    hipFree(device_output);
    hipFree(device_s);
}


// main calculation function
// public to the Machine learning part in the project
void CudaModel::calculate(double up, double down, double rate, double* array_s,
        double* array_output, int length){
    // length check
    if(length > MAXLENGTH){
        printf("error: ---length larger than system max!\n");
        return;
    }
    device_u = up;
    device_d = down;
    device_r = rate;
    hipMemcpy(device_s, array_s, length*sizeof(double),hipMemcpyHostToDevice);
    device_p = (1.0 + rate - down) / (up - down);
    device_q = 1.0 - device_p;

    // setup final stage, calculate all the leaf nodes
    int total_calc = length * (MAXLEVEL+1);
    int block_n = (total_calc + THREAD_PER_BLOCK -1) / THREAD_PER_BLOCK;
    kernelFinalStage<<<block_n, THREAD_PER_BLOCK>>>(length, MAXLEVEL, strike,
            device_buf, device_u, device_d, device_s);
    hipDeviceSynchronize();

    // calculate all nodes above leave
    for(int level = MAXLEVEL-1; level >= 0; level--){
        total_calc = length * (level+1);
        block_n = (total_calc + THREAD_PER_BLOCK -1) / THREAD_PER_BLOCK;
        kernelCalc<<<block_n, THREAD_PER_BLOCK>>>(level, MAXLEVEL, length,
                device_p, device_q, device_r, device_buf);
        hipDeviceSynchronize();
    }

    // output result
    kernelOutput<<<1, length>>>(device_output, device_buf, length, MAXLEVEL);
    hipDeviceSynchronize();
    hipMemcpy(array_output, device_output,
            length*sizeof(double), hipMemcpyDeviceToHost);
    return;
}
