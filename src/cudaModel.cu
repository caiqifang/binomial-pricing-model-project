#include "hip/hip_runtime.h"
#include <algorithm>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>
#include <stdio.h>

#include "cudaModel.h"

#define MAXLEVEL 600
// single buffer array size
#define BUFSIZE (MAXLEVEL+1)*(MAXLEVEL+2)/2
#define MAXLENGTH 10

#define THREAD_PER_BLOCK 128
/*
  This file is the implementation of parallel binomial model
   */

////////// KERNEL /////////////////////////
// In this project, we are calculating the call price
__device__ __inline__ double getValue(double strike, double stock){
        double value = stock - strike;
        if(value > 0.0)
            return value;
        return 0.0;
}
// kernel calculate inline


__global__ void kernelFinalStage(int length, double strike,
                            ){
        int index = blockDim.x * blockIdx.x + threadIdx.x;
        if( index >= length * (MAXLEVEL+1))
            return; // output bound



}


__global__ void kernel_calc(long double * device_data, bool * flag){
        // check flag
        int idx = blockDim.x * blockIdx.x + threadIdx.x;
        // calculate data

        // write result


}
//////////// HOST ///////////////////////////
// initialize the buffer and related structure
CudaModel::CudaModel(double s){
        hipMalloc(&device_p, sizeof(double)*MAXLENGTH);
        hipMalloc(&device_q, sizeof(double)*MAXLENGTH);
        hipMalloc(&device_buf, sizeof(double)*MAXLENGTH*BUFSIZE);
        hipMalloc(&device_u, sizeof(double)*MAXLENGTH);
        hipMalloc(&device_d, sizeof(double)*MAXLENGTH);
        hipMalloc(&device_s, sizeof(double)*MAXLENGTH);
        strike = s;
}

CudaModel::~CudaModel(){
        hipFree(device_p);
        hipFree(device_q);
        hipFree(device_buf);
        hipFree(device_u);
        hipFree(device_d);
        hipFree(device_s);
}

void CudaModel::calculate(double* array_u, double* array_d, double* array_s,
                          double* array_output, int length){
    if(length > MAXLENGTH){
        printf("error: ---length larger than system max!\n");
        return;
    }

    hipMemcpy(device_u, array_u, length*sizeof(double),hipMemcpyHostToDevice);
    hipMemcpy(device_d, array_d, length*sizeof(double),hipMemcpyHostToDevice);
    hipMemcpy(device_s, array_s, length*sizeof(double),hipMemcpyHostToDevice);
    for(int c = 0; c < length; c++){
        double p = (1.0 - array_d[c]) / (array_u[c] - array_d[c]);
        double q = 1.0 - p;
        array_p[c] = p;
        array_q[c] = q;
    }
    hipMemcpy(device_p, array_p, length*sizeof(double),hipMemcpyHostToDevice);
    hipMemcpy(device_q, array_q, length*sizeof(double),hipMemcpyHostToDevice);
        // setup final stage
        int total_calc = length * (MAXLEVEL+1);
        int block_n = (total_calc + THREAD_PER_BLOCK -1) / THREAD_PER_BLOCK;
        kernelFinalStage<<<block_n, THREAD_PER_BLOCK>>>( );
        // parallel by level
        for(int level = MAXLEVEL-1; level >= 0; level--){


        }
        // output result

}
