#include "hip/hip_runtime.h"
#include <algorithm>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>
#include <stdio.h>

#include "cudaModel.h"

#define MAXLEVEL 600
// single buffer array size
#define BUFSIZE (MAXLEVEL+1)*(MAXLEVEL+2)/2
#define MAXLENGTH 10

#define THREAD_PER_BLOCK 128
/*
   This file is the implementation of parallel binomial model
 */

////////// KERNEL /////////////////////////
// In this project, we are calculating the call price
__device__ __inline__ double getValue(double strike, double stock){
    double value = stock - strike;
    if(value > 0.0)
        return value;
    return 0.0;
}

__device__ __inline__ int getIndex(int length, int maxL, int level){
    return length*((maxL+1+level+2)*(maxL-level)/2);
}
// kernel calculate inline

__global__ void kernelFinalStage(int length, int maxL,  double strike,
        double* device_buf, double* device_u, double* device_d, double* device_s){
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    int size = maxL+1;
    if( index >= length * size)
        return; // output bound
    int element = index / size;
    double rank = __int2double_rn(index % size);
    double stock = device_s[element]; // s0
    double up = device_u[element];
    double down = device_d[element];
    double maxLevel = __int2double_rn(maxL);
    stock = stock * pow(up, maxLevel-rank) * pow(down, rank);
    device_buf[index] = getValue(strike, stock);
    return;
}

__global__ void kernelCalc(int level, int maxL, int length,
        double* device_p, double* device_q, double* device_buf){
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    int size = level + 1;
    if( index >= length * size)
        return; // output bound
    // WATCH!!
    int curr_start = getIndex( length, maxL, level);
    int prev_start = getIndex( length, maxL, level+1);
    int element = index / size;
    int rank = index % size;
    double p = device_p[element];
    double q = device_q[element];
    int head = prev_start + element*(size+1) + rank;
    int tail = head + 1;
    device_buf[curr_start+index] = (p*device_buf[head] + q*device_buf[tail]);
}

__global__ void kernelOutput(){
    int index = blockDim.x * blockIdx.x + threadIdx.x;
        // TODO
}
//////////// HOST ///////////////////////////
// initialize the buffer and related structure
CudaModel::CudaModel(double s){
    hipMalloc(&device_p, sizeof(double)*MAXLENGTH);
    hipMalloc(&device_q, sizeof(double)*MAXLENGTH);
    hipMalloc(&device_buf, sizeof(double)*MAXLENGTH*BUFSIZE);
    hipMalloc(&device_output, sizeof(double)*MAXLENGTH);
    hipMalloc(&device_u, sizeof(double)*MAXLENGTH);
    hipMalloc(&device_d, sizeof(double)*MAXLENGTH);
    hipMalloc(&device_s, sizeof(double)*MAXLENGTH);
    strike = s;
}

CudaModel::~CudaModel(){
    hipFree(device_p);
    hipFree(device_q);
    hipFree(device_buf);
    hipFree(device_output);
    hipFree(device_u);
    hipFree(device_d);
    hipFree(device_s);
}

void CudaModel::calculate(double* array_u, double* array_d, double* array_s,
        double* array_output, int length){
    if(length > MAXLENGTH){
        printf("error: ---length larger than system max!\n");
        return;
    }

    hipMemcpy(device_u, array_u, length*sizeof(double),hipMemcpyHostToDevice);
    hipMemcpy(device_d, array_d, length*sizeof(double),hipMemcpyHostToDevice);
    hipMemcpy(device_s, array_s, length*sizeof(double),hipMemcpyHostToDevice);
    for(int c = 0; c < length; c++){
        double p = (1.0 - array_d[c]) / (array_u[c] - array_d[c]);
        double q = 1.0 - p;
        array_p[c] = p;
        array_q[c] = q;
    }
    hipMemcpy(device_p, array_p, length*sizeof(double),hipMemcpyHostToDevice);
    hipMemcpy(device_q, array_q, length*sizeof(double),hipMemcpyHostToDevice);
    // setup final stage
    int total_calc = length * (MAXLEVEL+1);
    int block_n = (total_calc + THREAD_PER_BLOCK -1) / THREAD_PER_BLOCK;
    kernelFinalStage<<<block_n, THREAD_PER_BLOCK>>>(length, MAXLEVEL, strike,
            device_buf, device_u, device_d, device_s);
    hipDeviceSynchronize();
    // parallel by level
    for(int level = MAXLEVEL-1; level >= 0; level--){
        total_calc = length * (level+1);
        block_n = (total_calc + THREAD_PER_BLOCK -1) / THREAD_PER_BLOCK;
        kernelCalc<<<block_n, THREAD_PER_BLOCK>>>(level, MAXLEVEL,
                length, device_p, device_q, device_buf);
        hipDeviceSynchronize();
    }
    // output result
    kernelOutput<<<1, length>>>(); // TODO
    hipDeviceSynchronize();
    hipMemcpy(array_output, device_output,
            length*sizeof(double), hipMemcpyDeviceToHost);
    return;
}
